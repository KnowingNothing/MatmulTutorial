#include "hip/hip_runtime.h"
#include "common.h"
// copy async

// nvcc -arch=sm_90a -std=c++17 -I ../../include/ -lcuda ldgmem_ldsmem_v0.cu -o test

const int SM_LODA_BYTES = 128/8;

template <typename DType, int BLOCKM, int BLOCKN, int NUM_THREADS>
__global__ void naive_matrix_ldsm(DType* source, int M, int N, DType* dummy_out) {
    __shared__ DType smem[BLOCKM*BLOCKN];
    const int VEC_LEN = SM_LODA_BYTES / sizeof(DType);
    const int VEC_REPEAT = BLOCKN / VEC_LEN;
    const int THREAD_N = VEC_REPEAT;
    const int THREAD_M = NUM_THREADS / THREAD_N;
    const int ROW_REPEAT = BLOCKM / THREAD_M;
    static_assert(BLOCKN % VEC_LEN == 0);
    static_assert(NUM_THREADS % THREAD_N == 0);
    static_assert(ROW_REPEAT * THREAD_M == BLOCKM);

    int mo = blockIdx.x * BLOCKM;
    int mi = threadIdx.x / THREAD_N;
    int ni = threadIdx.x % THREAD_N;
    int4* ld_source = reinterpret_cast<int4*>(source);
    int4* ld_smem = reinterpret_cast<int4*>(smem);
    for (int no = 0; no < N; no += BLOCKN) {
        for (int row_repeat = 0; row_repeat < ROW_REPEAT; ++row_repeat) {
            int m = mo + row_repeat * THREAD_M + mi;
            int n = no + ni * VEC_LEN;
            int idx = m * N + n;
            int sm = row_repeat * THREAD_M + mi;
            int sn = ni * VEC_LEN;
            int sm_idx = sm * BLOCKN + sn;
            ld_smem[sm_idx / VEC_LEN] = ld_source[idx / VEC_LEN];
        }
        __syncthreads();
        for (int row_repeat = 0; row_repeat < ROW_REPEAT; ++row_repeat) {
            int m = mo + row_repeat * THREAD_M + mi;
            int n = no + ni * VEC_LEN;
            int idx = m * N + n;
            int sm = row_repeat * THREAD_M + mi;
            int sn = ni * VEC_LEN;
            int sm_idx = sm * BLOCKN + sn;
            for (int i = 0; i < VEC_LEN; ++i) {
                dummy_out[idx + i] = smem[sm_idx + i] + DType(1);
            }
        }
    }
}


template<typename DType>
void cpu_dummy(DType* source, DType* dummy_out, int M, int N) {
    for (int m = 0; m < M; ++m) {
        for (int n = 0; n < N; ++n) {
            dummy_out[m * N + n] = (DType)((float)source[m * N + n] + (float)DType(1));
        }
    }
}


int main(int argc, char** argv) {
    const int M = 1024;
    const int N = 1024;
    using DType = half;
    const int BLOCKM = 128;
    const int BLOCKN = 128;
    const int NUM_THREADS = 128;
    std::vector<int> shape{M, N};
    auto A = alloc_cpu_tensor<DType>(shape);
    random_fill(A, shape);
    auto B = alloc_cpu_tensor<DType>(shape);
    auto golden = alloc_cpu_tensor<DType>(shape);

    GPUTimer gpu_timer;

    auto dA = alloc_gpu_tensor<DType>(shape);
    auto dB = alloc_gpu_tensor<DType>(shape);
    gpu_timer.sync_all();
    gpu_timer.tick();
    copy_to_gpu_async(A, dA, shape);
    dim3 block(NUM_THREADS);
    dim3 grid(ceil_div(M, BLOCKM));
    naive_matrix_ldsm<DType, BLOCKM, BLOCKN, NUM_THREADS><<<grid, block>>>(dA, M, N, dB);
    copy_to_cpu_async(B, dB, shape);
    gpu_timer.tick();
    gpu_timer.sync_all();
    std::cout << "GPU naive done! Use " << gpu_timer.report_last_ms() << " ms.\n";

    std::cout << "Calculating golden...\n";
    cpu_dummy(A, golden, M, N);
    assert_allclose(B, golden, shape, 1e-5, /*dump=*/false);
    std::cout << "Correct!\n";

    
    return 0;
}