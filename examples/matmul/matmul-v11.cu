// 2 mma + pipeline + double threading + smem crosswise + ldmatrix + split K

// A100 PCIE 80GB
// Setting to 4 stages.
// Setting to 2x threading.
// Test performance using shape M=5376, N=5376, K=2048
// Running cost of CUDA kernel is 1.3483ms
// TFLOPS: 87.7994

// 3090
// Setting to 4 stages.
// Testing iters = 200.
// Setting to 2x threading.
// Test performance using shape M=5376, N=5376, K=2048
// Running cost of CUDA kernel is 1.99332ms
// TFLOPS: 59.3883

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

const int MI = 128;
const int NI = 128;
const int KI = 32;
const int MII = 64;
const int NII = 64;
const int KII = 16;
const int wmmaM = 16;
const int wmmaN = 16;
// const int wmmaK = 16;

__device__ __forceinline__ void loadSmemA(half *smem, half *A, int M, int K,
                                          int ko) {
  // load 128 * 32
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int tid = tz * 128 + ty * 32 + tx;
#pragma unroll
  for (int i = 0; i < 2; ++i) {
    int logic_row = i * 64 + tid / 4;
    int logic_col = tid % 4 * 8;
    int row = i * 32 + tid / 8;
    int col = tid % 8 * 8;
    col = col ^ (((row & 3) << 3));
    void *ptr = (void *)(smem + row * 64 + col);
    uint32_t smem_ptr;

    asm("{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 "
        "%0, smem_ptr; }\n"
        : "=r"(smem_ptr)
        : "l"(ptr));

    asm volatile("cp.async.cg.shared.global [%0], [%1], %2;\n" ::"r"(smem_ptr),
                 "l"(&A[(by * 128 + logic_row) * K + (ko * KI + logic_col)]),
                 "n"(16));
  }
}

__device__ __forceinline__ void loadSmemB(half *smem, half *B, int N, int K,
                                          int ko) {
  // load 128 * 32
  int bx = blockIdx.x;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int tid = tz * 128 + ty * 32 + tx;
#pragma unroll
  for (int i = 0; i < 2; ++i) {
    int logic_row = i * 64 + tid / 4;
    int logic_col = tid % 4 * 8;
    int row = i * 32 + tid / 8;
    int col = tid / 4 % 2 * 32 + tid % 4 * 8;
    col = col ^ (((row & 3) << 3));
    void *ptr = (void *)(smem + row * 64 + col);
    uint32_t smem_ptr;

    asm("{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 "
        "%0, smem_ptr; }\n"
        : "=r"(smem_ptr)
        : "l"(ptr));

    asm volatile("cp.async.cg.shared.global [%0], [%1], %2;\n" ::"r"(smem_ptr),
                 "l"(&B[(bx * 128 + logic_row) * K + (ko * KI + logic_col)]),
                 "n"(16));
  }
}

__device__ __forceinline__ void storeSmemC(half *C, half *smem, int M, int N) {
  // store 128 * 128
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int tid = tz * 128 + ty * 32 + tx;
  for (int i = 0; i < 64; ++i) {
    int row = i * 2 + tid / 128;
    int col = tid % 128;
    (C[(by * 128 + row) * N + bx * 128 + col]) =
        (half)(smem[row * 256 + col * 2] + smem[row * 256 + col * 2 + 1]);
  }
}

__device__ __forceinline__ void loadFragA(unsigned int *frag, half *smem) {
  // frag: [j, k]: [2, 2]
  // load 64x16
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;

#pragma unroll
  for (int i = 0; i < 4; ++i) {
    int row = tz * 64 + i * 16 + tx / 16 * 8 + tx % 8;
    int col = ty / 2 * KII + tx / 8 % 2 * 8;
    col = row % 2 * 32 + col;
    row = row / 2;
    col = col ^ (((row & 3) << 3));
    void *ptr = (void *)(smem + row * 64 + col);
    uint32_t smem_ptr;
    asm("{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 "
        "%0, smem_ptr; }\n"
        : "=r"(smem_ptr)
        : "l"(ptr));
    asm volatile(
        "ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n"
        : "=r"(frag[i * 4 + 0]), "=r"(frag[i * 4 + 1]), "=r"(frag[i * 4 + 2]),
          "=r"(frag[i * 4 + 3])
        : "r"(smem_ptr));
  }
}

__device__ __forceinline__ void loadFragB(unsigned int *frag, half *smem) {
  // frag: [j, k]: []
  // load 32x16
  int tx = threadIdx.x;
  int ty = threadIdx.y;
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    int row = ty % 2 * 64 + i * 16 + tx / 16 * 8 + tx % 8;
    int col = ty / 2 * KII + tx / 8 % 2 * 8;
    col = row % 2 * 32 + col;
    row = row / 2;
    col = col ^ (((row & 3) << 3));
    void *ptr = (void *)(smem + row * 64 + col);
    uint32_t smem_ptr;
    asm("{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 "
        "%0, smem_ptr; }\n"
        : "=r"(smem_ptr)
        : "l"(ptr));
    asm volatile(
        "ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n"
        : "=r"(frag[i * 4 + 0]), "=r"(frag[i * 4 + 1]), "=r"(frag[i * 4 + 2]),
          "=r"(frag[i * 4 + 3])
        : "r"(smem_ptr));
  }
}

__device__ __forceinline__ void storeAccum(half *ptr, float *frag) {
  // frag [r, c, _]: [2, 2, 2]
  // store 64x64
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  // smem view is [128x2x128]
#pragma unroll
  for (int i = 0; i < 4; ++i) {
#pragma unroll
    for (int j = 0; j < 4; ++j) {
      for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 2; ++c) {
          int row = tz * 64 + i * 16 + r * 8 + tx / 4;
          int col = ty % 2 * 64 + j * 16 + c * 8 + tx % 4 * 2;
          ptr[row * 256 + col * 2 + ty / 2] =
              (half)frag[i * 32 + j * 8 + r * 4 + c * 2 + 0];
          ptr[row * 256 + (col + 1) * 2 + ty / 2] =
              (half)frag[i * 32 + j * 8 + r * 4 + c * 2 + 1];
        }
      }
    }
  }
}

__device__ __forceinline__ void mmaSync(unsigned int *fragA,
                                        unsigned int *fragB, float *accum) {

  asm volatile("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
               "{%0,  %1,  %2,  %3},"
               "{%4,  %5,  %6,  %7},"
               "{%8,  %9},"
               "{%10, %11, %12, %13};\n"
               : "=f"(accum[0]), "=f"(accum[1]), "=f"(accum[4]), "=f"(accum[5])
               : "r"(fragA[0]), "r"(fragA[2]), "r"(fragA[1]), "r"(fragA[3]),
                 "r"(fragB[0]), "r"(fragB[1]), "f"(accum[0]), "f"(accum[1]),
                 "f"(accum[4]), "f"(accum[5]));

  asm volatile("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
               "{%0,  %1,  %2,  %3},"
               "{%4,  %5,  %6,  %7},"
               "{%8,  %9},"
               "{%10, %11, %12, %13};\n"
               : "=f"(accum[2]), "=f"(accum[3]), "=f"(accum[6]), "=f"(accum[7])
               : "r"(fragA[0]), "r"(fragA[2]), "r"(fragA[1]), "r"(fragA[3]),
                 "r"(fragB[2]), "r"(fragB[3]), "f"(accum[2]), "f"(accum[3]),
                 "f"(accum[6]), "f"(accum[7]));
}

__global__ void matmul(half *A, half *B, half *C, int M, int N, int K) {
  // A is row-major
  // B is col-major
  // 128 threads [x, y, z] = [32, 4, 2]
  // threadblock mma: 128x128x32
  // warp mma: 64x64x16
  extern __shared__ uint8_t shared_storage[];
  half *SA1 = reinterpret_cast<half *>(shared_storage);
  half *SA2 = SA1 + MI * KI;
  half *SA3 = SA2 + MI * KI;
  half *SA4 = SA3 + MI * KI;
  half *SB1 = SA4 + MI * KI;
  half *SB2 = SB1 + NI * KI;
  half *SB3 = SB2 + NI * KI;
  half *SB4 = SB3 + NI * KI;
  half *SC = reinterpret_cast<half *>(shared_storage);

  unsigned int FragA1[4 * 4]; // [4, 4]
  unsigned int FragB1[4 * 4]; // [4, 4]
                              //   unsigned int FragA2[4 * 4];      // [4, 4]
                              //   unsigned int FragB2[4 * 4];      // [4, 4]

  unsigned int *FragA[] = {FragA1};
  unsigned int *FragB[] = {FragB1};

  float Accum[4 * 4 * 8] = {0.0}; // [4, 4, 8]

  // prologue
  loadSmemA(SA1, A, M, K, 0);
  loadSmemB(SB1, B, N, K, 0);
  asm volatile("cp.async.commit_group;\n" ::);

  loadSmemA(SA2, A, M, K, 1);
  loadSmemB(SB2, B, N, K, 1);
  asm volatile("cp.async.commit_group;\n" ::);

  loadSmemA(SA3, A, M, K, 2);
  loadSmemB(SB3, B, N, K, 2);
  asm volatile("cp.async.commit_group;\n" ::);

  for (int ko = 0; ko < K / KI - 4; ko += 4) {

    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();

    if (ko + 3 < K / KI) {
      loadSmemA(SA4, A, M, K, ko + 3);
      loadSmemB(SB4, B, N, K, ko + 3);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    // for (int ki = 0; ki < KI / KII; ki += 1) {
    // 64x64x16 mma for each warp
    loadFragA(FragA[0], SA1);
    loadFragB(FragB[0], SB1);
#pragma unroll
    for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
      for (int mii = 0; mii < MII / wmmaM; mii += 1) {
        // 16x16x16 for each wmma
        mmaSync(&FragA[0][mii * 4], &FragB[0][nii * 4],
                &Accum[mii * 32 + nii * 8]);
      }
    }
    // }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();

    if (ko + 4 < K / KI) {
      loadSmemA(SA1, A, M, K, ko + 4);
      loadSmemB(SB1, B, N, K, ko + 4);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    // for (int ki = 0; ki < KI / KII; ki += 1) {
    // 64x64x16 mma for each warp
    loadFragA(FragA[0], SA2);
    loadFragB(FragB[0], SB2);
#pragma unroll
    for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
      for (int mii = 0; mii < MII / wmmaM; mii += 1) {
        // 16x16x16 for each wmma
        mmaSync(&FragA[0][mii * 4], &FragB[0][nii * 4],
                &Accum[mii * 32 + nii * 8]);
      }
    }
    // }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();

    if (ko + 5 < K / KI) {
      loadSmemA(SA2, A, M, K, ko + 5);
      loadSmemB(SB2, B, N, K, ko + 5);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    // for (int ki = 0; ki < KI / KII; ki += 1) {
    // 64x64x16 mma for each warp
    loadFragA(FragA[0], SA3);
    loadFragB(FragB[0], SB3);
#pragma unroll
    for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
      for (int mii = 0; mii < MII / wmmaM; mii += 1) {
        // 16x16x16 for each wmma
        mmaSync(&FragA[0][mii * 4], &FragB[0][nii * 4],
                &Accum[mii * 32 + nii * 8]);
      }
    }
    // }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();

    if (ko + 6 < K / KI) {
      loadSmemA(SA3, A, M, K, ko + 6);
      loadSmemB(SB3, B, N, K, ko + 6);
    }
    // for (int ki = 0; ki < KI / KII; ki += 1) {
    // 64x64x16 mma for each warp
    loadFragA(FragA[0], SA4);
    loadFragB(FragB[0], SB4);
#pragma unroll
    for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
      for (int mii = 0; mii < MII / wmmaM; mii += 1) {
        // 16x16x16 for each wmma
        mmaSync(&FragA[0][mii * 4], &FragB[0][nii * 4],
                &Accum[mii * 32 + nii * 8]);
      }
    }

    // }
  }

  // the last 4 iterations
  {

    int ko = (K / KI / 4 - 1) * 4;
    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();
    if (ko + 3 < K / KI) {
      loadSmemA(SA4, A, M, K, ko + 3);
      loadSmemB(SB4, B, N, K, ko + 3);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    // for (int ki = 0; ki < KI / KII; ki += 1) {
    // 64x64x16 mma for each warp
    loadFragA(FragA[0], SA1);
    loadFragB(FragB[0], SB1);
#pragma unroll
    for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
      for (int mii = 0; mii < MII / wmmaM; mii += 1) {
        // 16x16x16 for each wmma
        mmaSync(&FragA[0][mii * 4], &FragB[0][nii * 4],
                &Accum[mii * 32 + nii * 8]);
      }
    }
    // }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();
    if (ko + 4 < K / KI) {
      loadSmemA(SA1, A, M, K, ko + 4);
      loadSmemB(SB1, B, N, K, ko + 4);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    // for (int ki = 0; ki < KI / KII; ki += 1) {
    // 64x64x16 mma for each warp
    loadFragA(FragA[0], SA2);
    loadFragB(FragB[0], SB2);
#pragma unroll
    for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
      for (int mii = 0; mii < MII / wmmaM; mii += 1) {
        // 16x16x16 for each wmma
        mmaSync(&FragA[0][mii * 4], &FragB[0][nii * 4],
                &Accum[mii * 32 + nii * 8]);
      }
    }
    // }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(1));
    __syncthreads();
    if (ko + 5 < K / KI) {
      loadSmemA(SA2, A, M, K, ko + 5);
      loadSmemB(SB2, B, N, K, ko + 5);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    // for (int ki = 0; ki < KI / KII; ki += 1) {
    // 64x64x16 mma for each warp
    loadFragA(FragA[0], SA3);
    loadFragB(FragB[0], SB3);
#pragma unroll
    for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
      for (int mii = 0; mii < MII / wmmaM; mii += 1) {
        // 16x16x16 for each wmma
        mmaSync(&FragA[0][mii * 4], &FragB[0][nii * 4],
                &Accum[mii * 32 + nii * 8]);
      }
    }
    // }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(0));
    __syncthreads();
    if (ko + 6 < K / KI) {
      loadSmemA(SA3, A, M, K, ko + 6);
      loadSmemB(SB3, B, N, K, ko + 6);
    }
    // for (int ki = 0; ki < KI / KII; ki += 1) {
    // 64x64x16 mma for each warp
    loadFragA(FragA[0], SA4);
    loadFragB(FragB[0], SB4);
#pragma unroll
    for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
      for (int mii = 0; mii < MII / wmmaM; mii += 1) {
        // 16x16x16 for each wmma
        mmaSync(&FragA[0][mii * 4], &FragB[0][nii * 4],
                &Accum[mii * 32 + nii * 8]);
      }
    }
    // }
  }
  storeAccum(SC, Accum);
  __syncthreads();
  storeSmemC(C, SC, M, N);
}