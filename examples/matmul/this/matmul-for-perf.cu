// 2 mma + pipeline + smem crosswise + partial ldmatrix + no unroll + serpentine
// + ptx opt

// A100 PCIE 80GB

// 3090

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

const int MI = 128;
const int NI = 128;
const int KI = 32;
const int MII = 64;
const int NII = 64;
const int KII = 16;
const int wmmaM = 16;
const int wmmaN = 16;
// const int wmmaK = 16;

#define MIN(a, b) (a) < (b) ? (a) : (b)

__device__ __forceinline__ void loadSmemA(half *smem, half *A, int M, int K,
                                          int ko) {
  // load 128 * 32
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int tid = tz * 64 + ty * 32 + tx;
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    int logic_row = i * 32 + tid / 4;
    int logic_col = tid % 4 * 8;
    int row = i * 16 + tid / 8;
    int col = tid % 8 * 8;
    col = col ^ (((row & 3) << 3));
    void *ptr = (void *)(smem + row * 64 + col);
    uint32_t smem_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(ptr));

    asm volatile(
        "cp.async.cg.shared.global.L2::128B [%0], [%1], %2, %3;\n" ::"r"(
            smem_ptr),
        "l"(&A[(by * 128 + logic_row) * K + (ko * KI + logic_col)]), "n"(16),
        "r"(16));

    // uint32_t smem_ptr;

    // asm("{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 "
    //     "%0, smem_ptr; }\n"
    //     : "=r"(smem_ptr)
    //     : "l"(ptr));

    // asm volatile("cp.async.cg.shared.global [%0], [%1], %2;\n"
    // ::"r"(smem_ptr),
    //              "l"(&A[(by * 128 + logic_row) * K + (ko * KI + logic_col)]),
    //              "n"(16));
  }
}

__device__ __forceinline__ void predLoadSmemA(half *smem, half *A, int M, int K,
                                              int ko, bool pred_guard) {
  // load 128 * 32
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int tid = tz * 64 + ty * 32 + tx;
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    int logic_row = i * 32 + tid / 4;
    int logic_col = tid % 4 * 8;
    int row = i * 16 + tid / 8;
    int col = tid % 8 * 8;
    col = col ^ (((row & 3) << 3));
    void *ptr = (void *)(smem + row * 64 + col);
    uint32_t smem_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(ptr));

    asm volatile("{\n"
                 " .reg .pred p;\n"
                 " setp.ne.b32 p, %0, 0;\n"
                 " @p cp.async.cg.shared.global.L2::128B [%1], [%2], %3;\n"
                 "}\n" ::"r"((int)pred_guard),
                 "r"(smem_ptr),
                 "l"(&A[(by * 128 + logic_row) * K + (ko * KI + logic_col)]),
                 "n"(16));
  }
}

__device__ __forceinline__ void loadSmemB(half *smem, half *B, int N, int K,
                                          int ko) {
  // load 128 * 32
  int bx = blockIdx.x;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int tid = tz * 64 + ty * 32 + tx;
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    int logic_row = i * 32 + tid / 4;
    int logic_col = tid % 4 * 8;
    int row = i * 16 + tid / 8;
    int col = tid / 4 % 2 * 32 + tid % 4 * 8;
    col = col ^ (((row & 3) << 3));
    void *ptr = (void *)(smem + row * 64 + col);
    uint32_t smem_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(ptr));

    asm volatile(
        "cp.async.cg.shared.global.L2::128B [%0], [%1], %2, %3;\n" ::"r"(
            smem_ptr),
        "l"(&B[(bx * 128 + logic_row) * K + (ko * KI + logic_col)]), "n"(16),
        "r"(16));
  }
}

__device__ __forceinline__ void predLoadSmemB(half *smem, half *B, int N, int K,
                                              int ko, bool pred_guard) {
  // load 128 * 32
  int bx = blockIdx.x;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int tid = tz * 64 + ty * 32 + tx;
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    int logic_row = i * 32 + tid / 4;
    int logic_col = tid % 4 * 8;
    int row = i * 16 + tid / 8;
    int col = tid / 4 % 2 * 32 + tid % 4 * 8;
    col = col ^ (((row & 3) << 3));
    void *ptr = (void *)(smem + row * 64 + col);
    uint32_t smem_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(ptr));

    asm volatile("{\n"
                 " .reg .pred p;\n"
                 " setp.ne.b32 p, %0, 0;\n"
                 " @p cp.async.cg.shared.global.L2::128B [%1], [%2], %3;\n"
                 "}\n" ::"r"((int)pred_guard),
                 "r"(smem_ptr),
                 "l"(&B[(bx * 128 + logic_row) * K + (ko * KI + logic_col)]),
                 "n"(16));
  }
}

union Float4 {
  float4 f4;
  float2 f22[2];
};

__device__ __forceinline__ void storeSmemC(half *C, float *smem, int M, int N) {
  // load 128 * 128
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int tid = tz * 64 + ty * 32 + tx;
  for (int i = 0; i < 128; ++i) {
    int row = i;
    int col = tid;
    int scol = col ^ ((row & 3) << 3);
    (C[(by * 128 + row) * N + bx * 128 + col]) = (half)smem[row * 128 + scol];
  }
}

__device__ __forceinline__ void loadFragA(unsigned int *frag, half *smem,
                                          int ki) {
  // frag: [j, k]: [2, 2]
  // load 64x16
  int tx = threadIdx.x;
  int tz = threadIdx.z;

  //   load 16x16 at a time
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    int row = tz * 64 + i * 16 + tx / 16 * 8 + tx % 8;
    int col = ki * KII + tx / 8 % 2 * 8;
    col = row % 2 * 32 + col;
    row = row / 2;
    col = col ^ (((row & 3) << 3));
    void *ptr = (void *)(smem + row * 64 + col);
    // uint32_t smem_ptr;
    // asm("{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64"
    //     "%0, smem_ptr; }\n"
    //     : "=r"(smem_ptr)
    //     : "l"(ptr));
    uint32_t smem_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(ptr));
    asm volatile(
        "ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n"
        : "=r"(frag[i * 4 + 0]), "=r"(frag[i * 4 + 1]), "=r"(frag[i * 4 + 2]),
          "=r"(frag[i * 4 + 3])
        : "r"(smem_ptr));
  }
}

__device__ __forceinline__ void loadFragB(unsigned int *frag, half *smem,
                                          int ki) {
  // frag: [j, k]: []
  // load 64x16
  int tx = threadIdx.x;
  int ty = threadIdx.y;

// load 16x16 at a time
#pragma unroll
  for (int i = 0; i < 4; ++i) {
    int row = ty * 64 + i * 16 + tx / 16 * 8 + tx % 8;
    int col = ki * KII + tx / 8 % 2 * 8;
    col = row % 2 * 32 + col;
    row = row / 2;
    col = col ^ (((row & 3) << 3));
    void *ptr = (void *)(smem + row * 64 + col);
    // uint32_t smem_ptr;
    // asm("{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 "
    //     "%0, smem_ptr; }\n"
    //     : "=r"(smem_ptr)
    //     : "l"(ptr));
    uint32_t smem_ptr = static_cast<uint32_t>(__cvta_generic_to_shared(ptr));
    asm volatile(
        "ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n"
        : "=r"(frag[i * 4 + 0]), "=r"(frag[i * 4 + 1]), "=r"(frag[i * 4 + 2]),
          "=r"(frag[i * 4 + 3])
        : "r"(smem_ptr));
  }
}

__device__ __forceinline__ void storeAccum(float *ptr, float *frag) {
  // frag [r, c, _]: [2, 2, 2]
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  // smem view is [128x128]
#pragma unroll
  for (int i = 0; i < 4; ++i) {
#pragma unroll
    for (int j = 0; j < 4; ++j) {
      for (int r = 0; r < 2; ++r) {
        for (int c = 0; c < 2; ++c) {
          int row = tz * 64 + i * 16 + r * 8 + tx / 4;
          int col = ty * 64 + j * 16 + c * 8 + tx % 4 * 2;
          int scol = col ^ ((row & 3) << 3);
          ptr[row * 128 + scol] = frag[i * 32 + j * 8 + r * 4 + c * 2 + 0];
          ptr[row * 128 + (scol + 1)] =
              frag[i * 32 + j * 8 + r * 4 + c * 2 + 1];
        }
      }
    }
  }
}

template <int BM, int BN, int WM, int WN, int IM, int IN, int TX, int TY,
          int TZ>
struct Epilogue {
  static constexpr int WarpRepeatM = WM / IM;
  static constexpr int WarpRepeatN = WN / IN;
  static constexpr int TXY = TX * TY;
  static constexpr int storeElements = 16;
  static constexpr int storeRowThreads = BN / storeElements;
  int Bx = blockIdx.x;
  int By = blockIdx.y;
  int Bz = blockIdx.z;
  int Tx = threadIdx.x;
  int Ty = threadIdx.y;
  int Tz = threadIdx.z;

  __device__ __forceinline__ Epilogue(half *gmem_ptr, float *smem_ptr,
                                      float *frag_ptr, int M, int N,
                                      float alpha, float beta)
      : gmem_ptr(gmem_ptr), smem_ptr(smem_ptr), frag_ptr(frag_ptr), M(M), N(N),
        alpha(alpha), beta(beta) {
    store_smem_row_init = Tz * WM + Tx / 4;
    store_smem_col_init = Ty * WN + Tx % 4 * 2;
    int tid = Tz * TXY + Ty * TX + Tx;
    read_smem_row_init = tid / storeRowThreads;
    read_smem_col_init = (tid % storeRowThreads) * storeElements;
    store_smem_crosswise_factor = ((Tx / 4) & 3);
    read_smem_crosswise_factor = read_smem_row_init & 3;
  }

  __device__ __forceinline__ void operator()() {
    int store_smem_row = store_smem_row_init;
    int store_smem_col = store_smem_col_init;
    int read_smem_row = read_smem_row_init;
    int read_smem_col = read_smem_col_init;
    int store_gmem_row = By * BM + read_smem_row;
    int store_gmem_col = Bx * BN + read_smem_col;
    // store_smem_col = store_smem_col ^ (store_smem_crosswise_factor << 3);
    // int store_smem_minor_stride = store_smem_crosswise_factor & 1 ? -4 : 4;
    // int store_smem_minor_strides_1[] = {-8, 24, -8, 24};
    // int store_smem_minor_strides_2[] = {8, 8, 8, 8};
    // int* store_smem_major_strides = store_smem_crosswise_factor & 2 ?
    // store_smem_minor_strides_1 : store_smem_minor_strides_2; read_smem_col =
    // read_smem_col ^ (read_smem_crosswise_factor << 3); int
    // read_smem_minor_stride =
    float2 *store_smem_ptr = reinterpret_cast<float2 *>(
        smem_ptr + store_smem_row * BN + store_smem_col);
    float2 *read_frag_ptr = reinterpret_cast<float2 *>(frag_ptr);
    float4 *read_smem_ptr = reinterpret_cast<float4 *>(
        smem_ptr + read_smem_row * BN + read_smem_col);
    uint4 *store_gmem_ptr = reinterpret_cast<uint4 *>(
        gmem_ptr + store_gmem_row * N + store_gmem_col);
    for (int wm = 0; wm < WarpRepeatM; ++wm) {
      // store frag to smem
      float2 *tmp_store_smem_ptr = store_smem_ptr;
      float2 *tmp_read_frag_ptr = read_frag_ptr;
      for (int wn = 0; wn < WarpRepeatN; ++wn) {
        // thread row 0, col 0, 2 elements;
        tmp_store_smem_ptr[0] = tmp_read_frag_ptr[0];
        // thread row 0, col 1, 2 elements;
        tmp_store_smem_ptr[4] = tmp_read_frag_ptr[1];
        // thread row 1, col 0, 2 elements;
        tmp_store_smem_ptr[4 * BN] = tmp_read_frag_ptr[2];
        // thread row 1, col 1, 2 elements;
        tmp_store_smem_ptr[4 * BN + 4] = tmp_read_frag_ptr[3];
        tmp_store_smem_ptr += 8;
        tmp_read_frag_ptr += 4;
      }
      store_smem_ptr += 8 * BN;
      read_frag_ptr += 16;

      __syncthreads();

      // store smem to gmem
      float4 *tmp_read_smem_ptr = read_smem_ptr;
      uint4 *tmp_store_gmem_ptr = store_gmem_ptr;
      for (int i = 0; i < 2;
           ++i) { // how to better calculate this extend statically?
        // read
        float4 smem_read_values[4];
        float2 mul_values[8];
        half2 cast_values[8];
        uint4 *to_store = reinterpret_cast<uint4 *>(cast_values);
        // 1st 4 elements
        smem_read_values[0] = tmp_read_smem_ptr[0];
        // 2nd 4 elements
        smem_read_values[1] = tmp_read_smem_ptr[1];
        // 3rd 4 elements
        smem_read_values[2] = tmp_read_smem_ptr[2];
        // 4th 4 elements
        smem_read_values[3] = tmp_read_smem_ptr[3];
        // mul
        mul_values[0].x = smem_read_values[0].x * alpha;
        mul_values[0].y = smem_read_values[0].y * alpha;
        mul_values[1].x = smem_read_values[0].z * alpha;
        mul_values[1].y = smem_read_values[0].w * alpha;
        mul_values[2].x = smem_read_values[1].x * alpha;
        mul_values[2].y = smem_read_values[1].y * alpha;
        mul_values[3].x = smem_read_values[1].z * alpha;
        mul_values[3].y = smem_read_values[1].w * alpha;
        // cast
        cast_values[0] = __float22half2_rn(mul_values[0]);
        cast_values[1] = __float22half2_rn(mul_values[1]);
        cast_values[2] = __float22half2_rn(mul_values[2]);
        cast_values[3] = __float22half2_rn(mul_values[3]);
        // mul
        mul_values[4].x = smem_read_values[2].x * alpha;
        mul_values[4].y = smem_read_values[2].y * alpha;
        mul_values[5].x = smem_read_values[2].z * alpha;
        mul_values[5].y = smem_read_values[2].w * alpha;
        mul_values[6].x = smem_read_values[3].x * alpha;
        mul_values[6].y = smem_read_values[3].y * alpha;
        mul_values[7].x = smem_read_values[3].z * alpha;
        mul_values[7].y = smem_read_values[3].w * alpha;
        // store
        tmp_store_gmem_ptr[0] = to_store[0];
        // cast
        cast_values[4] = __float22half2_rn(mul_values[4]);
        cast_values[5] = __float22half2_rn(mul_values[5]);
        cast_values[6] = __float22half2_rn(mul_values[6]);
        cast_values[7] = __float22half2_rn(mul_values[7]);
        // store
        tmp_store_gmem_ptr[1] = to_store[1];
        // update ptr
        tmp_read_smem_ptr += 16 * BN;
        tmp_store_gmem_ptr += 8 * N;
      }
      read_smem_ptr += 4 * BN;
      store_gmem_ptr += 2 * N;
    }
  }

  half *gmem_ptr;
  float *smem_ptr;
  float *frag_ptr;
  int M;
  int N;
  float alpha;
  float beta;
  int store_smem_row_init;
  int store_smem_col_init;
  int read_smem_row_init;
  int read_smem_col_init;
  int store_smem_crosswise_factor;
  int read_smem_crosswise_factor;
};

__device__ __forceinline__ void epilogue(half *gmem_ptr, float *smem_ptr,
                                         float *frag_ptr, int M, int N,
                                         float alpha, float beta) {
  // frag [r, c, _]: [2, 2, 2]
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  // smem view is [128x128]
#pragma unroll
  for (int i = 0; i < 4; ++i) {
#pragma unroll
    for (int j = 0; j < 4; ++j) {
      int row = tz * 64 + i * 16 + 0 * 8 + tx / 4;
      for (int r = 0; r < 2; ++r) {
        int scol = ty * 64 + j * 16 + 0 * 8 + tx % 4 * 2;
        for (int c = 0; c < 2; ++c) {
          // scol = scol ^ ((row & 3) << 3);
          float2 *ptr = reinterpret_cast<float2 *>(smem_ptr + row * 128 + scol);
          float2 *frag = reinterpret_cast<float2 *>(frag_ptr + i * 32 + j * 8 +
                                                    r * 4 + c * 2 + 0);
          *ptr = *frag;
          scol += 8;
        }
        row += 8;
      }
    }
    __syncthreads();

    int p = tx / 2;
    int q = tx % 2;
    int srow = tz * 64 + i * 16 + p;
    int scol = ty * 64 + 0 * 16 + q * 8;
    for (int j = 0; j < 4; ++j) {
      half2 a[4];
      float4 value[2];
      // int scol = col ^ ((srow & 3) << 3);
      float4 *ptr = reinterpret_cast<float4 *>(smem_ptr + srow * 128 + scol);
      float2 *asFloat2 = reinterpret_cast<float2 *>(value);
      float *asFloat = reinterpret_cast<float *>(value + j * 2);
      for (int x = 0; x < 8; ++x) {
        asFloat[x] *= alpha;
        asFloat[x] += beta;
      }
      uint4 *asUInt4 = reinterpret_cast<uint4 *>(a);
      value[0] = *ptr;
      value[1] = *(ptr + 1);
      a[0] = __float22half2_rn(asFloat2[0]);
      a[1] = __float22half2_rn(asFloat2[1]);
      a[2] = __float22half2_rn(asFloat2[2]);
      a[3] = __float22half2_rn(asFloat2[3]);
      uint4 *dst = reinterpret_cast<uint4 *>(gmem_ptr + (by * 128 + srow) * N +
                                             bx * 128 + scol);
      *dst = *asUInt4;
      scol += 16;
    }
  }
}

__device__ __forceinline__ void mmaSync(unsigned int *fragA,
                                        unsigned int *fragB, float *accum) {
  asm volatile("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
               "{%0,  %1,  %2,  %3},"
               "{%4,  %5,  %6,  %7},"
               "{%8,  %9},"
               "{%10, %11, %12, %13};\n"
               : "=f"(accum[0]), "=f"(accum[1]), "=f"(accum[4]), "=f"(accum[5])
               : "r"(fragA[0]), "r"(fragA[2]), "r"(fragA[1]), "r"(fragA[3]),
                 "r"(fragB[0]), "r"(fragB[1]), "f"(accum[0]), "f"(accum[1]),
                 "f"(accum[4]), "f"(accum[5]));

  asm volatile("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
               "{%0,  %1,  %2,  %3},"
               "{%4,  %5,  %6,  %7},"
               "{%8,  %9},"
               "{%10, %11, %12, %13};\n"
               : "=f"(accum[2]), "=f"(accum[3]), "=f"(accum[6]), "=f"(accum[7])
               : "r"(fragA[0]), "r"(fragA[2]), "r"(fragA[1]), "r"(fragA[3]),
                 "r"(fragB[2]), "r"(fragB[3]), "f"(accum[2]), "f"(accum[3]),
                 "f"(accum[6]), "f"(accum[7]));
}

__global__ void matmul(half *A, half *B, half *C, int M, int N, int K,
                       float alpha, float beta) {
  // A is row-major
  // B is col-major
  // 128 threads [x, y, z] = [32, 2, 2]
  // threadblock mma: 128x128x32
  // warp mma: 64x64x16
  extern __shared__ uint8_t shared_storage[];
  half *SA1 = reinterpret_cast<half *>(shared_storage);
  half *SA2 = SA1 + MI * KI;
  half *SA3 = SA2 + MI * KI;
  half *SA4 = SA3 + MI * KI;
  half *SB1 = SA4 + MI * KI;
  half *SB2 = SB1 + NI * KI;
  half *SB3 = SB2 + NI * KI;
  half *SB4 = SB3 + NI * KI;
  half *SA[] = {SA1, SA2, SA3, SA4};
  half *SB[] = {SB1, SB2, SB3, SB4};
  float *SC = reinterpret_cast<float *>(shared_storage);

  unsigned int FragA[2][4 * 4]; // [4, 4]
  unsigned int FragB[2][4 * 4]; // [4, 4]

  float Accum[4 * 4 * 8] = {0.0}; // [4, 4, 8]

  // FragIteratorA frag_iter_A(SA[0]);

  // prologue
  for (int i = 0; i < 3; ++i) {
    loadSmemA(SA[i], A, M, K, i);
    loadSmemB(SB[i], B, N, K, i);
    asm volatile("cp.async.commit_group;\n" ::);
  }

  asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
  __syncthreads();

  loadFragA(FragA[0], SA[(0) % 4], 0);
  // frag_iter_A(FragA[0]);
  // ++frag_iter_A;
  loadFragB(FragB[0], SB[(0) % 4], 0);

  for (int ko = 0; ko < K / KI; ko += 1) {

    // 64x64x16 mma for each warp
    loadFragA(FragA[1], SA[(ko) % 4], 1);
    // frag_iter_A(FragA[1]);
    // ++frag_iter_A;
    loadFragB(FragB[1], SB[(ko) % 4], 1);
#pragma unroll
    for (int mii = 0; mii < MII / wmmaM; mii += 1) {
#pragma unroll
      for (int nii = 0; nii < NII / wmmaN; nii += 1) {
        // 16x16x16 for each wmma
        int n = (mii & 1) ? NII / wmmaN - 1 - nii : nii;
        mmaSync(&FragA[0][mii * 4], &FragB[0][n * 4], &Accum[mii * 32 + n * 8]);
      }
    }

    // if (ko + 3 < K / KI) {
    //     loadSmemA(SA[(ko+3)%4], A, M, K, ko + 3);
    //     loadSmemB(SB[(ko+3)%4], B, N, K, ko + 3);
    //     asm volatile("cp.async.commit_group;\n" ::);
    // }
    bool pred_guard = ko + 3 < K / KI;
    predLoadSmemA(SA[(ko + 3) % 4], A, M, K, ko + 3, pred_guard);
    predLoadSmemB(SB[(ko + 3) % 4], B, N, K, ko + 3, pred_guard);
    asm volatile("cp.async.commit_group;\n" ::);

    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();

    // 64x64x16 mma for each warp
    loadFragA(FragA[0], SA[(ko + 1) % 4], 0);
    // frag_iter_A.set(SA[(ko+1)%4]);
    // frag_iter_A(FragA[0]);
    // ++frag_iter_A;
    loadFragB(FragB[0], SB[(ko + 1) % 4], 0);
#pragma unroll
    for (int mii = 0; mii < MII / wmmaM; mii += 1) {
#pragma unroll
      for (int nii = 0; nii < NII / wmmaN; nii += 1) {
        // 16x16x16 for each wmma
        int n = (mii & 1) ? NII / wmmaN - 1 - nii : nii;
        mmaSync(&FragA[1][mii * 4], &FragB[1][n * 4], &Accum[mii * 32 + n * 8]);
      }
    }
  }

  storeAccum(SC, Accum);
  __syncthreads();
  storeSmemC(C, SC, M, N);
  // epilogue(C, SC, Accum, M, N, alpha, beta);
  // Epilogue<128, 128, 64, 64, 16, 16, 32, 2, 2> epi(C, SC, Accum, M, N, alpha,
  //                                                  beta);
  // epi();
}