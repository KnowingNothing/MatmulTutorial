// 2 mma + pipeline + double threading + smem crosswise + ldmatrix

// A100 PCIE 80GB
// Setting to 4 stages.
// Setting to 2x threading.
// Test performance using shape M=5376, N=5376, K=2048
// Running cost of CUDA kernel is 1.10695ms
// TFLOPS: 106.942

// 3090
// Setting to 4 stages.
// Testing iters = 200.
// Setting to 2x threading.
// Test performance using shape M=5376, N=5376, K=2048
// Running cost of CUDA kernel is 1.93203ms
// TFLOPS: 61.2723

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

const int MI = 128;
const int NI = 128;
const int KI = 32;
const int MII = 64;
const int NII = 32;
const int KII = 16;
const int wmmaM = 16;
const int wmmaN = 16;
// const int wmmaK = 16;

__device__ __forceinline__ void loadSmemA(half *smem, half *A, int M, int K,
                                          int ko) {
  // load 128 * 32
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int tid = tz * 128 + ty * 32 + tx;
#pragma unroll
  for (int i = 0; i < 2; ++i) {
    int logic_row = i * 64 + tid / 4;
    int logic_col = tid % 4 * 8;
    int row = i * 32 + tid / 8;
    int col = tid % 8 * 8;
    col = col ^ (((row & 3) << 3));
    void *ptr = (void *)(smem + row * 64 + col);
    uint32_t smem_ptr;

    asm("{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 "
        "%0, smem_ptr; }\n"
        : "=r"(smem_ptr)
        : "l"(ptr));

    asm volatile("cp.async.cg.shared.global [%0], [%1], %2;\n" ::"r"(smem_ptr),
                 "l"(&A[(by * 128 + logic_row) * K + (ko * KI + logic_col)]),
                 "n"(16));
  }
}

__device__ __forceinline__ void loadSmemB(half *smem, half *B, int N, int K,
                                          int ko) {
  // load 128 * 32
  int bx = blockIdx.x;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int tid = tz * 128 + ty * 32 + tx;
#pragma unroll
  for (int i = 0; i < 2; ++i) {
    int logic_row = i * 64 + tid / 4;
    int logic_col = tid % 4 * 8;
    int row = i * 32 + tid / 8;
    int col = tid / 4 % 2 * 32 + tid % 4 * 8;
    col = col ^ (((row & 3) << 3));
    void *ptr = (void *)(smem + row * 64 + col);
    uint32_t smem_ptr;

    asm("{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 "
        "%0, smem_ptr; }\n"
        : "=r"(smem_ptr)
        : "l"(ptr));

    asm volatile("cp.async.cg.shared.global [%0], [%1], %2;\n" ::"r"(smem_ptr),
                 "l"(&B[(bx * 128 + logic_row) * K + (ko * KI + logic_col)]),
                 "n"(16));
  }
}

__device__ __forceinline__ void loadSmemC(float *smem, half *C, int M, int N) {
  // load 128 * 128
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int tid = tz * 128 + ty * 32 + tx;
  for (int i = 0; i < 64; ++i) {
    int row = i * 2 + tid / 128;
    int col = tid % 128;
    smem[row * 128 + col] = (float)(C[(by * 128 + row) * N + bx * 128 + col]);
  }
}

__device__ __forceinline__ void storeSmemC(half *C, float *smem, int M, int N) {
  // load 128 * 128
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int tid = tz * 128 + ty * 32 + tx;
  for (int i = 0; i < 64; ++i) {
    int row = i * 2 + tid / 128;
    int col = tid % 128;
    (C[(by * 128 + row) * N + bx * 128 + col]) = (half)smem[row * 128 + col];
  }
}

__device__ __forceinline__ void loadFragA(unsigned int *frag, half *smem,
                                          int ki) {
  // frag: [j, k]: [2, 2]
  // load 64x16
  int tx = threadIdx.x;
  int tz = threadIdx.z;

  // for (int i = 0; i < 4; ++i) {
  //   for (int j = 0; j < 2; ++j) {
  //     for (int k = 0; k < 2; ++k) {
  //       int row = tz * 64 + i * 16 + j * 8 + tx / 4;
  //       int col = ki * KII + k * 8 + tx % 4 * 2;
  //       col = row % 2 * 32 + col;
  //       row = row / 2;
  //       col = col ^ ((row & 3) << 3);
  //       unsigned int *ptr =
  //           reinterpret_cast<unsigned int *>(smem + row * 64 + col);
  //       frag[i * 4 + j * 2 + k] = ptr[0];
  //     }
  //   }
  // }

#pragma unroll
  for (int i = 0; i < 4; ++i) {
    int row = tz * 64 + i * 16 + tx / 16 * 8 + tx % 8;
    int col = ki * KII + tx / 8 % 2 * 8;
    col = row % 2 * 32 + col;
    row = row / 2;
    col = col ^ (((row & 3) << 3));
    void *ptr = (void *)(smem + row * 64 + col);
    uint32_t smem_ptr;
    asm("{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 "
        "%0, smem_ptr; }\n"
        : "=r"(smem_ptr)
        : "l"(ptr));
    asm volatile(
        "ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n"
        : "=r"(frag[i * 4 + 0]), "=r"(frag[i * 4 + 1]), "=r"(frag[i * 4 + 2]),
          "=r"(frag[i * 4 + 3])
        : "r"(smem_ptr));
  }
}

__device__ __forceinline__ void loadFragB(unsigned int *frag, half *smem,
                                          int ki) {
  // frag: [j, k]: []
  // load 32x16
  // int tx = threadIdx.x;
  // int ty = threadIdx.y;
  // for (int i = 0; i < 2; ++i) {
  //   for (int j = 0; j < 2; ++j) {
  //     for (int k = 0; k < 2; ++k) {
  //       int row = ty * 32 + i * 16 + j * 8 + tx / 4;
  //       int col = ki * KII + k * 8 + tx % 4 * 2;
  //       col = row % 2 * 32 + col;
  //       row = row / 2;
  //       col = col ^ ((row & 3) << 3);
  //       unsigned int *ptr =
  //           reinterpret_cast<unsigned int *>(smem + row * 64 + col);
  //       frag[i * 4 + j * 2 + k] = ptr[0];
  //     }
  //   }
  // }
  int tx = threadIdx.x;
  int ty = threadIdx.y;
#pragma unroll
  for (int i = 0; i < 2; ++i) {
    int row = ty * 32 + i * 16 + tx / 16 * 8 + tx % 8;
    int col = ki * KII + tx / 8 % 2 * 8;
    col = row % 2 * 32 + col;
    row = row / 2;
    col = col ^ (((row & 3) << 3));
    void *ptr = (void *)(smem + row * 64 + col);
    uint32_t smem_ptr;
    asm("{ .reg .u64 smem_ptr; cvta.to.shared.u64 smem_ptr, %1; cvt.u32.u64 "
        "%0, smem_ptr; }\n"
        : "=r"(smem_ptr)
        : "l"(ptr));
    asm volatile(
        "ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n"
        : "=r"(frag[i * 4 + 0]), "=r"(frag[i * 4 + 1]), "=r"(frag[i * 4 + 2]),
          "=r"(frag[i * 4 + 3])
        : "r"(smem_ptr));
  }
}

__device__ __forceinline__ void storeAccum(float *ptr, float *frag) {
  // frag [r, c, _]: [2, 2, 2]
  // store 64x32
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
  int row = tz * 64 + tx / 4;
  int col = ty * 32 + tx % 4 * 2;
  // float *dst = ptr + row / 16 * (8 * 16 * 16) + col / 16 * (16 * 16) + row %
  // 16 * 16 + col % 16;
  float *dst = ptr + row * 128 + col;
#pragma unroll
  for (int i = 0; i < 4; ++i) {
#pragma unroll
    for (int j = 0; j < 2; ++j) {
      dst[0] = frag[i * 16 + j * 8 + 0 * 4 + 0 * 2];
      dst[1] = frag[i * 16 + j * 8 + 0 * 4 + 0 * 2 + 1];

      dst[0 + 8] = frag[i * 16 + j * 8 + 0 * 4 + 1 * 2];
      dst[1 + 8] = frag[i * 16 + j * 8 + 0 * 4 + 1 * 2 + 1];

      dst[0 + 8 * 128] = frag[i * 16 + j * 8 + 1 * 4 + 0 * 2];
      dst[1 + 8 * 128] = frag[i * 16 + j * 8 + 1 * 4 + 0 * 2 + 1];

      dst[0 + 8 * 128 + 8] = frag[i * 16 + j * 8 + 1 * 4 + 1 * 2];
      dst[1 + 8 * 128 + 8] = frag[i * 16 + j * 8 + 1 * 4 + 1 * 2 + 1];

      dst += 16;
    }
    dst += 16 * 128 - 16 * 2;
  }
}

__device__ __forceinline__ void mmaSync(unsigned int *fragA,
                                        unsigned int *fragB, float *accum) {
  asm volatile("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
               "{%0,  %1,  %2,  %3},"
               "{%4,  %5,  %6,  %7},"
               "{%8,  %9},"
               "{%10, %11, %12, %13};\n"
               : "=f"(accum[0]), "=f"(accum[1]), "=f"(accum[4]), "=f"(accum[5])
               : "r"(fragA[0]), "r"(fragA[2]), "r"(fragA[1]), "r"(fragA[3]),
                 "r"(fragB[0]), "r"(fragB[1]), "f"(accum[0]), "f"(accum[1]),
                 "f"(accum[4]), "f"(accum[5]));

  asm volatile("mma.sync.aligned.m16n8k16.row.col.f32.f16.f16.f32 "
               "{%0,  %1,  %2,  %3},"
               "{%4,  %5,  %6,  %7},"
               "{%8,  %9},"
               "{%10, %11, %12, %13};\n"
               : "=f"(accum[2]), "=f"(accum[3]), "=f"(accum[6]), "=f"(accum[7])
               : "r"(fragA[0]), "r"(fragA[2]), "r"(fragA[1]), "r"(fragA[3]),
                 "r"(fragB[2]), "r"(fragB[3]), "f"(accum[2]), "f"(accum[3]),
                 "f"(accum[6]), "f"(accum[7]));
}

__global__ void matmul(half *A, half *B, half *C, int M, int N, int K, float alpha, float beta) {
  // A is row-major
  // B is col-major
  // 128 threads [x, y, z] = [32, 2, 2]
  // threadblock mma: 128x128x32
  // warp mma: 64x64x16
  extern __shared__ uint8_t shared_storage[];
  half *SA1 = reinterpret_cast<half *>(shared_storage);
  half *SA2 = SA1 + MI * KI;
  half *SA3 = SA2 + MI * KI;
  half *SA4 = SA3 + MI * KI;
  half *SB1 = SA4 + MI * KI;
  half *SB2 = SB1 + NI * KI;
  half *SB3 = SB2 + NI * KI;
  half *SB4 = SB3 + NI * KI;
  float *SC = reinterpret_cast<float *>(shared_storage);

  unsigned int FragA[4 * 4]; // [4, 4]
  unsigned int FragB[2 * 4]; // [2, 4]

  float Accum[4 * 2 * 8] = {0.0}; // [4, 2, 8]

  // prologue
  loadSmemA(SA1, A, M, K, 0);
  loadSmemB(SB1, B, N, K, 0);
  asm volatile("cp.async.commit_group;\n" ::);

  loadSmemA(SA2, A, M, K, 1);
  loadSmemB(SB2, B, N, K, 1);
  asm volatile("cp.async.commit_group;\n" ::);

  loadSmemA(SA3, A, M, K, 2);
  loadSmemB(SB3, B, N, K, 2);
  asm volatile("cp.async.commit_group;\n" ::);

  for (int ko = 0; ko < K / KI - 4; ko += 4) {
    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();
    if (ko + 3 < K / KI) {
      loadSmemA(SA4, A, M, K, ko + 3);
      loadSmemB(SB4, B, N, K, ko + 3);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    for (int ki = 0; ki < KI / KII; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA1, ki);
      loadFragB(FragB, SB1, ki);
#pragma unroll
      for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
        for (int mii = 0; mii < MII / wmmaM; mii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();
    if (ko + 4 < K / KI) {
      loadSmemA(SA1, A, M, K, ko + 4);
      loadSmemB(SB1, B, N, K, ko + 4);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    for (int ki = 0; ki < KI / KII; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA2, ki);
      loadFragB(FragB, SB2, ki);
#pragma unroll
      for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
        for (int mii = 0; mii < MII / wmmaM; mii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();
    if (ko + 5 < K / KI) {
      loadSmemA(SA2, A, M, K, ko + 5);
      loadSmemB(SB2, B, N, K, ko + 5);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    for (int ki = 0; ki < KI / KII; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA3, ki);
      loadFragB(FragB, SB3, ki);
#pragma unroll
      for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
        for (int mii = 0; mii < MII / wmmaM; mii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();
    if (ko + 6 < K / KI) {
      loadSmemA(SA3, A, M, K, ko + 6);
      loadSmemB(SB3, B, N, K, ko + 6);
    }
    for (int ki = 0; ki < KI / KII; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA4, ki);
      loadFragB(FragB, SB4, ki);
#pragma unroll
      for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
        for (int mii = 0; mii < MII / wmmaM; mii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }
  }

  // the last 4 iterations
  {
    int ko = (K / KI / 4 - 1) * 4;
    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();
    if (ko + 3 < K / KI) {
      loadSmemA(SA4, A, M, K, ko + 3);
      loadSmemB(SB4, B, N, K, ko + 3);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    for (int ki = 0; ki < KI / KII; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA1, ki);
      loadFragB(FragB, SB1, ki);
#pragma unroll
      for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
        for (int mii = 0; mii < MII / wmmaM; mii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(2));
    __syncthreads();
    if (ko + 4 < K / KI) {
      loadSmemA(SA1, A, M, K, ko + 4);
      loadSmemB(SB1, B, N, K, ko + 4);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    for (int ki = 0; ki < KI / KII; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA2, ki);
      loadFragB(FragB, SB2, ki);
#pragma unroll
      for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
        for (int mii = 0; mii < MII / wmmaM; mii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(1));
    __syncthreads();
    if (ko + 5 < K / KI) {
      loadSmemA(SA2, A, M, K, ko + 5);
      loadSmemB(SB2, B, N, K, ko + 5);
      asm volatile("cp.async.commit_group;\n" ::);
    }
    for (int ki = 0; ki < KI / KII; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA3, ki);
      loadFragB(FragB, SB3, ki);
#pragma unroll
      for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
        for (int mii = 0; mii < MII / wmmaM; mii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }

    asm volatile("cp.async.wait_group %0;\n" ::"n"(0));
    __syncthreads();
    if (ko + 6 < K / KI) {
      loadSmemA(SA3, A, M, K, ko + 6);
      loadSmemB(SB3, B, N, K, ko + 6);
    }
    for (int ki = 0; ki < KI / KII; ki += 1) {
      // 64x64x16 mma for each warp
      loadFragA(FragA, SA4, ki);
      loadFragB(FragB, SB4, ki);
#pragma unroll
      for (int nii = 0; nii < NII / wmmaN; nii += 1) {
#pragma unroll
        for (int mii = 0; mii < MII / wmmaM; mii += 1) {
          // 16x16x16 for each wmma
          mmaSync(&FragA[mii * 4], &FragB[nii * 4], &Accum[mii * 16 + nii * 8]);
        }
      }
    }
  }
  storeAccum(SC, Accum);
  __syncthreads();
  storeSmemC(C, SC, M, N);
}