#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int device;
    hipError_t cudaStatus;

    // Get the currently active device
    cudaStatus = hipGetDevice(&device);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipGetDevice failed!" << std::endl;
        return 1;
    }

    // Query the shared memory size per block
    int sharedMemPerSM, sharedMemPerBlock;
    cudaStatus = hipDeviceGetAttribute(&sharedMemPerSM, hipDeviceAttributeMaxSharedMemoryPerMultiprocessor, device);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceGetAttribute failed!" << std::endl;
        return 1;
    }
    cudaStatus = hipDeviceGetAttribute(&sharedMemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, device);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceGetAttribute failed!" << std::endl;
        return 1;
    }

    std::cout << "Shared memory available per sm: " << sharedMemPerSM << " bytes" << std::endl;
    std::cout << "Shared memory available per block: " << sharedMemPerBlock << " bytes" << std::endl;
    return 0;
}