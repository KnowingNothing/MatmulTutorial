// wmma + fake pipeline

// A100 PCIE 80GB
// Test performance using shape M=5376, N=5376, K=2048
// Running cost of CUDA kernel is 3.58903ms
// TFLOPS: 32.9838

// 3090
// Test performance using shape M=5376, N=5376, K=2048
// Running cost of CUDA kernel is 6.1813ms
// TFLOPS: 19.1513

#include <hip/hip_fp16.h>
#include <mma.h>
#include <hip/hip_runtime.h>

const int MI = 128;
const int NI = 128;
const int KI = 32;
const int MII = 64;
const int NII = 64;
const int KII = 16;
const int wmmaM = 16;
const int wmmaN = 16;
const int wmmaK = 16;

__device__ void loadSmemA(half *smem, half *A, int M, int K, int ko)
{
    // load 128 * 32
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    int tid = tz * 64 + ty * 32 + tx;
    for (int i = 0; i < 32; ++i)
    {
        int row = i * 4 + tid / 32;
        int col = tid % 32;
        // layout: [row_out, col_out, row_in, col_in] = [8, 2, 16, 16]
        smem[row / 16 * (2 * 16 * 16) + col / 16 * (16 * 16) + row % 16 * 16 + col % 16] = A[(by * 128 + row) * K + ko * KI + col];
    }
}

__device__ void loadSmemB(half *smem, half *B, int N, int K, int ko)
{
    // load 128 * 32
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    int tid = tz * 64 + ty * 32 + tx;
    for (int i = 0; i < 32; ++i)
    {
        int row = i * 4 + tid / 32;
        int col = tid % 32;
        // layout: [row_out, col_out, row_in, col_in] = [8, 2, 16, 16]
        smem[row / 16 * (2 * 16 * 16) + col / 16 * (16 * 16) + row % 16 * 16 + col % 16] = B[(bx * 128 + row) * K + ko * KI + col];
    }
}

__device__ void loadSmemC(float *smem, half *C, int M, int N)
{
    // load 128 * 128
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    int tid = tz * 64 + ty * 32 + tx;
    for (int i = 0; i < 128; ++i)
    {
        int row = i;
        int col = tid;
        // layout: [row_out, col_out, row_in, col_in] = [8, 8, 16, 16]
        smem[row / 16 * (8 * 16 * 16) + col / 16 * (16 * 16) + row % 16 * 16 + col % 16] = (float)(C[(by * 128 + row) * N + bx * 128 + col]);
    }
}

__device__ void storeSmemC(half *C, float *smem, int M, int N)
{
    // load 128 * 128
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    int tid = tz * 64 + ty * 32 + tx;
    for (int i = 0; i < 128; ++i)
    {
        int row = i;
        int col = tid;
        // layout: [row_out, col_out, row_in, col_in] = [8, 8, 16, 16]
        (C[(by * 128 + row) * N + bx * 128 + col]) = (half)smem[row / 16 * (8 * 16 * 16) + col / 16 * (16 * 16) + row % 16 * 16 + col % 16];
    }
}

__device__ void loadFragA(nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, wmmaM, wmmaN, wmmaK, half, nvcuda::wmma::row_major> *frag, half *smem, int ki)
{
    // load 64x16
    int tz = threadIdx.z;
    for (int i = 0; i < 4; ++i)
    {
        int row = tz * 64 + i * 16;
        int col = ki * KII;
        nvcuda::wmma::load_matrix_sync(frag[i], smem + row / 16 * (2 * 16 * 16) + col / 16 * (16 * 16), 16);
    }
}

__device__ void loadFragB(nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, wmmaM, wmmaN, wmmaK, half, nvcuda::wmma::col_major> *frag, half *smem, int ki)
{
    // load 64x16
    int ty = threadIdx.y;
    for (int i = 0; i < 4; ++i)
    {
        int row = ty * 64 + i * 16;
        int col = ki * KII;
        nvcuda::wmma::load_matrix_sync(frag[i], smem + row / 16 * (2 * 16 * 16) + col / 16 * (16 * 16), 16);
    }
}

__device__ void storeAccum(float *ptr, nvcuda::wmma::fragment<nvcuda::wmma::accumulator, wmmaM, wmmaN, wmmaK, float> *frag)
{
    // store 64x64
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    for (int i = 0; i < 4; ++i)
    {
        for (int j = 0; j < 4; ++j)
        {
            int row = tz * 64 + i * 16;
            int col = ty * 64 + j * 16;
            // laoyut: [8, 8, 16, 16]
            nvcuda::wmma::store_matrix_sync(ptr + row / 16 * (8 * 16 * 16) + col / 16 * (16 * 16), frag[i * 4 + j], 16, nvcuda::wmma::mem_row_major);
        }
    }
}

__global__ void matmul(half *A, half *B, half *C, int M, int N, int K)
{
    // A is row-major
    // B is col-major
    // 128 threads [x, y, z] = [32, 2, 2]
    // threadblock mma: 128x128x32
    // warp mma: 64x64x16
    extern __shared__ uint8_t shared_storage[];
    half *SA1 = reinterpret_cast<half *>(shared_storage);
    half *SA2 = SA1 + MI * KI;
    half *SA3 = SA2 + MI * KI;
    half *SA4 = SA3 + MI * KI;
    half *SB1 = SA4 + MI * KI;
    half *SB2 = SB1 + NI * KI;
    half *SB3 = SB2 + NI * KI;
    half *SB4 = SB3 + NI * KI;
    float *SC = reinterpret_cast<float *>(shared_storage);

    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, wmmaM, wmmaN, wmmaK, half, nvcuda::wmma::row_major> FragA[MII / wmmaM];
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, wmmaM, wmmaN, wmmaK, half, nvcuda::wmma::col_major> FragB[NII / wmmaN];
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, wmmaM, wmmaN, wmmaK, float> Accum[MII / wmmaM * NII / wmmaN];

    for (int mii = 0; mii < MII / wmmaM; mii += 1)
    {
        for (int nii = 0; nii < NII / wmmaN; nii += 1)
        {
            nvcuda::wmma::fill_fragment(Accum[mii * (NII / wmmaN) + nii], 0.0);
        }
    }

    // prologue
    loadSmemA(SA1, A, M, K, 0);
    loadSmemB(SB1, B, N, K, 0);

    loadSmemA(SA2, A, M, K, 1);
    loadSmemB(SB2, B, N, K, 1);

    loadSmemA(SA3, A, M, K, 2);
    loadSmemB(SB3, B, N, K, 2);

    for (int ko = 0; ko < K / KI; ko += 4)
    {
        __syncthreads();
        if (ko + 3 < K / KI)
        {
            loadSmemA(SA4, A, M, K, ko + 3);
            loadSmemB(SB4, B, N, K, ko + 3);
        }
        for (int ki = 0; ki < KI / KII; ki += 1)
        {
            // 64x64x16 mma for each warp
            loadFragA(FragA, SA1, ki);
            loadFragB(FragB, SB1, ki);
            for (int mii = 0; mii < MII / wmmaM; mii += 1)
            {
                for (int nii = 0; nii < NII / wmmaN; nii += 1)
                {
                    // 16x16x16 for each wmma
                    nvcuda::wmma::mma_sync(Accum[mii * (NII / wmmaN) + nii], FragA[mii], FragB[nii], Accum[mii * (NII / wmmaN) + nii]);
                }
            }
        }

        __syncthreads();
        if (ko + 4 < K / KI)
        {
            loadSmemA(SA1, A, M, K, ko + 4);
            loadSmemB(SB1, B, N, K, ko + 4);
        }
        for (int ki = 0; ki < KI / KII; ki += 1)
        {
            // 64x64x16 mma for each warp
            loadFragA(FragA, SA2, ki);
            loadFragB(FragB, SB2, ki);
            for (int mii = 0; mii < MII / wmmaM; mii += 1)
            {
                for (int nii = 0; nii < NII / wmmaN; nii += 1)
                {
                    // 16x16x16 for each wmma
                    nvcuda::wmma::mma_sync(Accum[mii * (NII / wmmaN) + nii], FragA[mii], FragB[nii], Accum[mii * (NII / wmmaN) + nii]);
                }
            }
        }

        __syncthreads();
        if (ko + 5 < K / KI)
        {
            loadSmemA(SA2, A, M, K, ko + 5);
            loadSmemB(SB2, B, N, K, ko + 5);
        }
        for (int ki = 0; ki < KI / KII; ki += 1)
        {
            // 64x64x16 mma for each warp
            loadFragA(FragA, SA3, ki);
            loadFragB(FragB, SB3, ki);
            for (int mii = 0; mii < MII / wmmaM; mii += 1)
            {
                for (int nii = 0; nii < NII / wmmaN; nii += 1)
                {
                    // 16x16x16 for each wmma
                    nvcuda::wmma::mma_sync(Accum[mii * (NII / wmmaN) + nii], FragA[mii], FragB[nii], Accum[mii * (NII / wmmaN) + nii]);
                }
            }
        }

        __syncthreads();
        if (ko + 6 < K / KI)
        {
            loadSmemA(SA3, A, M, K, ko + 6);
            loadSmemB(SB3, B, N, K, ko + 6);
        }
        for (int ki = 0; ki < KI / KII; ki += 1)
        {
            // 64x64x16 mma for each warp
            loadFragA(FragA, SA4, ki);
            loadFragB(FragB, SB4, ki);
            for (int mii = 0; mii < MII / wmmaM; mii += 1)
            {
                for (int nii = 0; nii < NII / wmmaN; nii += 1)
                {
                    // 16x16x16 for each wmma
                    nvcuda::wmma::mma_sync(Accum[mii * (NII / wmmaN) + nii], FragA[mii], FragB[nii], Accum[mii * (NII / wmmaN) + nii]);
                }
            }
        }
    }
    storeAccum(SC, Accum);
    __syncthreads();
    storeSmemC(C, SC, M, N);
}